#include "hip/hip_runtime.h"
#include "cudaHeader.cuh"

__global__ void roughGradient(float* imageData,float* gradientOut,float* directions,int pitch,int width,int height,int sep)
{
    int _u = threadIdx.x + blockIdx.x * blockDim.x * sep;  //需要调整
    int v = threadIdx.y + blockIdx.y * blockDim.y;
    //一个线程展开为4,内存访问的为对齐和合并的

    for(int i=0;i<sep;++i){
        int u = _u + blockDim.x * i;
        if((u < 1 || u > (width - 2)) || (v < 1 || v > (height - 2)))
            continue;
        int centerIdx = v * pitch + u;
        int topIdx = (v - 1) * pitch + u;
        int btmIdx = (v + 1) * pitch + u;
        float h_grad = -1 * imageData[topIdx - 1] + 0 * imageData[topIdx] + 1 * imageData[topIdx+1] + \
                       -2 * imageData[centerIdx - 1] + 0 * imageData[centerIdx] + 2 * imageData[centerIdx+1] +\
                       -1 * imageData[btmIdx - 1] + 0 * imageData[btmIdx] + 1 * imageData[btmIdx+1] ;

        float v_grad = -1 * imageData[topIdx - 1] + -2 * imageData[topIdx] + -1 * imageData[topIdx+1] + \
                       0 * imageData[centerIdx - 1] + 0 * imageData[centerIdx] + 0 * imageData[centerIdx+1] +\
                       1 * imageData[btmIdx - 1] + 2* imageData[btmIdx] + 1 * imageData[btmIdx+1];

        gradientOut[v * width + u] = sqrtf(__powf(fabs(h_grad),2.) + __powf(fabs(v_grad),2.));
        //printf("uu[2]: %f",uu[2]);
        //float direction = atanf(fabs(v_grad) / fabs(h_grad)) / 3.1416 * 180.f + 90.f;
        float direction = fabs(atan2f(v_grad , h_grad) / 3.1416 * 180.f);
        directions[v * width + u] = direction;
    }

}


__global__ void GetGridHisto(float* gradientValIn,float* gradientDirectionIn,float* histoOut,\
                             int sep,int bins,int width,int height,int gridSize,int gridX)
{
    int _x = threadIdx.x + blockIdx.x * blockDim.x;
    int _y = threadIdx.y + blockIdx.y * blockDim.y;     //_x,_y对应grid的位置
    int _u = _x * gridSize;
    int _v = _y * gridSize;
    //每个thread对应一个grid,thread当前的位置对应于block所处的位置
    if(_u > (width - gridSize - 1) || _v > (height - gridSize - 1)) //不足一个grid大小,不进行统计
        return;

    int rootIdx = _y * gridX + _x;  //一个block水平方向有blockDim.x个grid,一张图像水平方向有gridDim.x个block
    //_u和_v分别为各个grid左上点的坐标
    //非合并访问(...)
    float* root_address = histoOut + rootIdx * bins;    //定位到相应grid存放bin值的内存
    for(int i=_u;i<_u+gridSize;++i){    //无需范围判断
        for(int j=_v;j<_v+gridSize;++j){
            int idx = j * width + i;
            float gradient = gradientValIn[idx];
            float angle = gradientDirectionIn[idx];
            int floor = int((angle  - 1e-5) / sep);
            int ceil = floor + 1;

            float floor_wei = (ceil * sep - angle) / sep * gradient;
            float ceil_wei = (angle - floor * sep) / sep * gradient;
            if(ceil >= bins)
                ceil = 0;
            atomicAdd((root_address + floor),floor_wei);    //统计值
            atomicAdd((root_address + ceil),ceil_wei);
        }
    }
}


__global__ void Hog(float* gridHistoIn,float* hogOut,int gridPerRaw,int gridPerCol,int blockPerRaw,int bins)
{
    //一个线程负责一个block的组建,读取四个grid的histo写入hogOut相对应的位置,无写入冲突
    int _u = threadIdx.x + blockIdx.x * blockDim.x;
    int _v = threadIdx.y + blockIdx.y * blockDim.y;
    if(_u > (gridPerRaw - 2) || _v > (gridPerCol - 2))    //最后一个grid不计数
        return;
    //_u和_v对应于grid序号
    //取出两个起始头,每一个头往后为(18)位数据,
    float* header1 = gridHistoIn + (_u + _v * gridPerRaw) * bins;
    float* header2 = gridHistoIn + (_u + (_v+1) * gridPerRaw) * bins;   //下一行
    float* dstheader = hogOut + (_u + _v * blockPerRaw) * 4 * bins; //blockPerRaw为每行的cell个数-1
    //header1+18位 + header2+18位
    for(int i=0;i<bins * 2;++i){
        *(dstheader+i) = *(header1 + i);
        *(dstheader+i + 2 * bins) = *(header2 + i);
    }
    //normalize...,起点dstheader向后(4) * bins

    float squ_sum=0;
    for(int k=0;k<4*bins;++k)
    {
        squ_sum += __powf(fabs(*(dstheader+k)),2.);
    }

    squ_sum = sqrtf(squ_sum + 1e-7);
    for(int k=0;k<4*bins;++k)
    {
        *(dstheader+k) /= squ_sum;
    }
}

void roughGradientExcuter(float* imageData,float* gradientOut,float* directions,int pitch,int width,int height,int sep,dim3 grid,dim3 block)
{
    roughGradient<<<grid,block>>>(imageData,gradientOut,directions,pitch,width,height,sep);
}


void HOGGridHistogram(float* gradientValIn,float* gradientDirectionIn,float* histoOut,int sep,int bins,\
                      int width,int height,int gridSize,int gridX,dim3 grid,dim3 block)
{
    GetGridHisto<<<grid,block>>>(gradientValIn,gradientDirectionIn,histoOut,sep,bins,width,height,gridSize,gridX);
}

void HOGDecriptor(float* gridHistoIn,float* hogOut,int gridPerRaw,int gridPerCol,int blockPerRaw,int bins,dim3 grid,dim3 block)
{
    Hog<<<grid,block>>>(gridHistoIn,hogOut,gridPerRaw,gridPerCol,blockPerRaw,bins);
}
